
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <sstream>
#include <fstream>
#include <numeric>
#include <stdlib.h>
//#include <cutil.h>
#include <vector>
#include <algorithm>
using namespace std;
#define REDUCE_BLOCK_SIZE 128

struct Matrix {
	Matrix() : elements(NULL), width(0), height(0), pitch(0) {}
	~Matrix() { if (elements) delete[] elements; }
	unsigned int width;
    unsigned int height;
    unsigned int pitch;
    float* elements;
};

__global__ void matrixMulKernel(float*, float*, float*, int, int, int, int);
__global__ void sigmoidKernel(float*, int);
__global__ void matrixAbsErrorKernel(float*, float*, float*, int, int);
__global__ void absErrorKernel(float*, float*, float*, int);
__global__ void updateParamsAbsErrorKernel(float*, float*, float*, float*, int, float);
__global__ void crossEntropyKernel(float*, float*, float*, int);
__global__ void reduceKernel(float*, float*, int);

inline static void InitializeMatrix(Matrix *mat, int x, int y, float val) {
	if (x > mat->width || y > mat->height) {
		throw ("invalid access - Initialize Matrix");
	}
	mat->elements[y * mat->width + x] = val;
}

inline static float Matrix_Element_Required(Matrix *mat, int x, int y)
{
	if (x > mat->width || y > mat->height) {
		throw ("invalid access - Matrix Element Required");
	}
	return mat->elements[y * mat->width + x];
}

static void AllocateMatrix(Matrix *mat, int height, int width)
{
	mat->elements = new float[height * width];
	mat->width = width;
	mat->height = height;
	for (int i = 0; i < mat->width; i++) {
		for (int j = 0; j < mat->height; j++) {
			InitializeMatrix(mat, i, j, 0.0f);
		}
	}
}

static void DisplayMatrix(Matrix &mat, bool force = false)
{
	std::cout << "Dim: " << mat.height << ", " << mat.width << "\n";
	if ((mat.width < 10 && mat.height < 10) || force)
	{
		for (int j = 0; j < mat.height; j++) {
			for (int i = 0; i < mat.width; i++) {
				std::cout << Matrix_Element_Required(&mat, i, j) << "\t";
			}
			std::cout << "\n";
		}
	}
	std::cout << std::endl;
}

static bool setup_data (string file_name, Matrix *X, Matrix *y) {

	ifstream s(file_name.c_str());
	//ifstream s(file_name);
	if (!s.is_open()) {
		//throw runtime_error(file_name + " doesn't exist");
		printf("The file does not exist\n");
	}

	int rows = 0;
	int cols = 0;
	string line;
	while (getline(s, line)) {
		// if we read first line, check how many columns
		if (rows++ == 0) {
			stringstream ss(line);

			while (ss.good()) {
				string substr;
				getline(ss, substr, ',');
				cols++;
			}
		}
	}
	std::cout << "Found " << rows << " rows with " << cols << " columns." << std::endl;
	s.clear() ;
	s.seekg(0, ios::beg);

	AllocateMatrix (X, rows - 1,cols - 2);
	AllocateMatrix (y, rows - 1, 1);

	// go to second line
	getline(s, line);
	int ya = 0;
	while (getline(s, line)) {
		stringstream ss(line);

		int xa = 0;
		while (ss.good()) {
			string substr;
			getline(ss, substr, ',');
			// first column is uninteresting
			// second column is target values
			if (xa == 1) {
				float val = atof(substr.c_str());
				InitializeMatrix(y, 0, ya, val);
			} else if (xa > 1) {
				float val = atof(substr.c_str());
				InitializeMatrix(X, (xa - 2), ya, val);
			}
			xa++;
		}
		ya++;
	}

	return true;
}

static void Normalize_Matrix_min_max(Matrix *m)
{
	for (int x = 0; x < m->width; ++x) {
		// calculate std for each column
		float min = Matrix_Element_Required(m, x, 0);
		float max = Matrix_Element_Required(m, x, 0);
		for (int y = 1; y < m->height; ++y) {
			float val = Matrix_Element_Required(m, x, y);
			if (val < min) {
				min = val;
			} else if (val > max) {
				max = val;
			}
		}

		for (int y = 0; y < m->height; ++y) {
			float val = Matrix_Element_Required(m, x, y);
			InitializeMatrix(m, x, y, (val - min) / max);
		}
	}
}

static void InitializeRandom(Matrix *mat, float LO, float HI)
{
	for (int i = 0; i < mat->width; ++i) {
		for (int j = 0; j < mat->height; ++j) {
			float r = LO + static_cast <float> (rand()) /( static_cast <float> (RAND_MAX/(HI-LO)));
			InitializeMatrix(mat, i, j, r);
		}
	}
}

static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}
#define SAFE_CALL(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

__global__ void matrixMulKernel(float *m1, float *m2, float *r, int m1w, int m2w, int rw, int rh)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if ((row < rh) && (col < rw)) {
		// dot product
		float accum = 0.0f;
		for (int c = 0; c < m1w; c++)
		{
			float v1 = m1[row * m1w + c];
			float v2 = m2[c * m2w + col];
			accum += (v1 *  v2);
		}

		r[row * rw + col] = accum;
	}
}

__global__ void sigmoidKernel(float *r, int m)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < m) {
		float val = r[index];
		r[index] = 1.0 / (1.0 + expf(-val));
	}
}

__global__ void matrixAbsErrorKernel(float *p, float *ys, float *r, int rw, int rh)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if ((row < rh) && (col < rw)) {
		float pval = p[row * rw + col];
		float ysval = ys[row * rw + col];

		float v = pval - ysval;
		r[row * rw + col] = v * v;
	}
}

__global__ void absErrorKernel(float *p, float *ys, float *r, int m)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < m) {
		float pval = p[index];
		float ysval = ys[index];

		float v = pval - ysval;
		r[index] = v * v;
	}
}

__global__ void updateParamsAbsErrorKernel(float *p, float *ys, float *th, float *xs, int m, float alpha)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < m) {
		float h = *p;
		float y = *ys;

		float x = xs[index];

		th[index] = th[index] - alpha * (h - y) * x;
	}
}

__global__ void crossEntropyKernel(float *p, float *ys, float *r, int m)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < m) {
		float pval = p[index];
		float ysval = ys[index];

		float ex = log1pf(expf(-ysval * pval));
		r[index] = ex;
	}
}

__global__ void reduceKernel(float * input, float * output, int len) {
    //@@ Load a segment of the input vector into shared memory
    __shared__ float partialSum[2 * REDUCE_BLOCK_SIZE];
    unsigned int t = threadIdx.x, start = 2 * blockIdx.x * REDUCE_BLOCK_SIZE;
    if (start + t < len)
       partialSum[t] = input[start + t];
    else
       partialSum[t] = 0;
    if (start + REDUCE_BLOCK_SIZE + t < len)
       partialSum[REDUCE_BLOCK_SIZE + t] = input[start + REDUCE_BLOCK_SIZE + t];
    else
       partialSum[REDUCE_BLOCK_SIZE + t] = 0;
    //@@ Traverse the reduction tree
    for (unsigned int stride = REDUCE_BLOCK_SIZE; stride >= 1; stride >>= 1) {
       __syncthreads();
       if (t < stride)
          partialSum[t] += partialSum[t+stride];
    }
    //@@ Write the computed sum of the block to the output vector at the
    //@@ correct index
    if (t == 0)
       output[blockIdx.x] = partialSum[0];
}

static void Logistic_Regression_CUDA(Matrix *X, Matrix *y, Matrix *Parameters, Matrix *Train_Parameters, int maxIterations, float alpha, vector<float> &cost_function)
{
	// put stuff into gpu
	float *gpu_X;
	float *gpu_y;

	float *gpu_prediction;

	float *gpu_params;
	float *gpu_abs_error;
	float *gpu_err_cost;

	float *gpu_predictions;
	Matrix predictions;
	AllocateMatrix(&predictions, y->height, y->width);

	Matrix absErrors;
	AllocateMatrix(&absErrors, y->height, y->width);

	float mean_error;
	float sum=0;
	int quantity = 1;

	int m = y->height;

	int numOutputElements;
	numOutputElements = m / (REDUCE_BLOCK_SIZE<<1);
	if (m % (REDUCE_BLOCK_SIZE<<1)) {
		numOutputElements++;
	}

	SAFE_CALL(hipMalloc((void**)&gpu_X, sizeof(float) * X->width * X->height));
	SAFE_CALL(hipMalloc((void**)&gpu_y, sizeof(float) * y->width * y->height));
	SAFE_CALL(hipMalloc((void**)&gpu_prediction, sizeof(float)));
	SAFE_CALL(hipMalloc((void**)&gpu_predictions, sizeof(float) * y->width * y->height));
	SAFE_CALL(hipMalloc((void**)&gpu_abs_error, sizeof(float) * y->width * y->height));
	SAFE_CALL(hipMalloc((void**)&gpu_params, sizeof(float) * Parameters->width * Parameters->height));
	SAFE_CALL(hipMalloc((void**)&gpu_err_cost, sizeof(float) * numOutputElements));

	SAFE_CALL(hipMemcpy(gpu_X, X->elements, sizeof(float) * X->width * X->height, hipMemcpyHostToDevice));
	SAFE_CALL(hipMemcpy(gpu_y, y->elements, sizeof(float) * y->width * y->height, hipMemcpyHostToDevice));
	SAFE_CALL(hipMemcpy(gpu_params, Parameters->elements, sizeof(float) * Parameters->width * Parameters->height, hipMemcpyHostToDevice));

	// invoke kernel
	static const int blockWidth = 16;
	static const int blockHeight = blockWidth;
	int numBlocksW = X->width / blockWidth;
	int numBlocksH = X->height / blockHeight;
	if (X->width % blockWidth) numBlocksW++;
	if (X->height % blockHeight) numBlocksH++;

	dim3 dimGrid(numBlocksW, numBlocksH);
	dim3 dimBlock(blockWidth, blockHeight);

	dim3 dimReduce((m - 1) / REDUCE_BLOCK_SIZE + 1);
	dim3 dimReduceBlock(REDUCE_BLOCK_SIZE);

	dim3 dimVectorGrid(((m - 1) / blockWidth * blockWidth) + 1);
	dim3 dimVectorBlock(blockWidth * blockWidth);

	float* error_accum = new float[numOutputElements];
	for (int iter = 0; iter < maxIterations; ++iter) {
		for (int i = 0; i < m; ++i) {
			matrixMulKernel<<<dimGrid, dimBlock>>>(&gpu_X[i * X->width], gpu_params, gpu_prediction, X->width, Parameters->width, 1, 1);
			sigmoidKernel<<<dimVectorGrid, dimVectorBlock>>>(gpu_prediction, 1);
			updateParamsAbsErrorKernel<<<dimVectorGrid, dimVectorBlock>>>(gpu_prediction, &gpu_y[i], gpu_params, &gpu_X[i * X->width], Parameters->height, alpha);
		}
		matrixMulKernel<<<dimGrid, dimBlock>>>(gpu_X, gpu_params, gpu_predictions, X->width, Parameters->width, predictions.width, predictions.height);
		sigmoidKernel<<<dimVectorGrid, dimVectorBlock>>>(gpu_predictions, m);


		// calculate error
		absErrorKernel<<<dimVectorGrid, dimVectorBlock>>>(gpu_predictions, gpu_y, gpu_abs_error, m);
		reduceKernel<<<dimReduce, dimReduceBlock>>>(gpu_abs_error, gpu_err_cost, m);
		SAFE_CALL(hipMemcpy(error_accum, gpu_err_cost, sizeof(float) * numOutputElements, hipMemcpyDeviceToHost));
		float g_sum = 0;
		for (int i = 0; i < numOutputElements; ++i)
		{
			g_sum += error_accum[i];
		}

		g_sum /= (2*m);

		cost_function.push_back(g_sum);
		sum += g_sum;
		quantity++;
		cout << g_sum << "\n";
	}

	mean_error = sum/quantity;
	printf("\n The mean error is %f\n", mean_error);
	cout << endl;

	delete[] error_accum;
	SAFE_CALL(hipFree(gpu_X));
	SAFE_CALL(hipFree(gpu_y));
	SAFE_CALL(hipFree(gpu_abs_error));
	SAFE_CALL(hipFree(gpu_prediction));
	SAFE_CALL(hipFree(gpu_predictions));
	SAFE_CALL(hipFree(gpu_params));
	SAFE_CALL(hipFree(gpu_err_cost));
}

int main(int argc, char *argv[])
{
	string input_file = "";
	cout << "Please enter a valid file to run test for logistic regression on CUDA:\n>";
	getline(cin, input_file);
 	cout << "You entered: " << input_file << endl << endl;
    Matrix X,y;
    setup_data (input_file, &X, &y);
    cout <<"\n The X - Squiggle Matrix." << endl;
    DisplayMatrix (X,true);
    cout <<"\n The y - Matrix." << endl;
    DisplayMatrix (y,true);

    Matrix Parameters, Train_Parameters;
    //Setup matrices with 1 as value initially
    AllocateMatrix(&Parameters, X.width, 1);
    AllocateMatrix(&Train_Parameters, X.width, 1);
    //Initialize with random +1 and -1 parameters.
    InitializeRandom(&Parameters, -1.0, 1.0);

    Normalize_Matrix_min_max(&X);

    vector<float> cost_function;

    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    //unsigned int timer;
    //CUT_SAFE_CALL(cutCreateTimer(&timer));
    
    //cutStartTimer(timer);
    hipEventRecord(start);
    Logistic_Regression_CUDA(&X, &y, &Parameters, &Train_Parameters, 150, 0.03, cost_function);
    //cutStopTimer(timer);
    hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	//printf("\nProcessing time: %f (ms)\n", cutGetTimerValue(timer));
	printf("\nProcessing time: %f (ms)\n", milliseconds);

    std::cout << "**********************done!********************* - CUDA Project - Sarthak - Vladislav" << std::endl;

	return 0;
}